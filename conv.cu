
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
using namespace std;
#define eps 1e-4

//每个thread负责output的一个pixel
__global__ void convolution2d(float *img, float *kernel, float* result, int n, int m, int kw, int kh, int out_n, int out_m, bool padding)
{
    int bx = blockIdx.x, by = blockIdx.y;
    int tx = threadIdx.x, ty = threadIdx.y;
    int x = bx * blockDim.x + tx;
    int y = by * blockDim.y + ty;
    int idx = y * out_m + x;
    //printf("%d %d %d %d %d %d\n", bx, by, tx, ty, x, y);
    if(idx < out_n * out_m){
        float ret = 0;
        for(int i = 0; i < kw; i++){
            for(int j = 0; j < kh; j++){
                //ret += img[(y + j) * m + (x + i)] * kernel[i * kh + j];
                //padding = same: (x,y) 为中心点，(x-kw/2, y-kh/2)为左上角第一个点
                //padding = valid: (x+kw/2, y+kh/2)为中心点, (x,y)为左上角第一个点
                int cur_x = 0, cur_y = 0;
                if(padding == true){
                    cur_x = x - kw / 2 + i;
                    cur_y = y - kh / 2 + j;
                }
                else{
                    cur_x = x + i;
                    cur_y = y + j;
                }
                if(cur_x >= 0 and cur_x < n and cur_y >= 0 and cur_y < m){
                    ret += img[cur_y * m + cur_x] * kernel[i * kh + j];
                }
            }
        }
        //printf("%d %d %d %f\n", x, y, idx, ret);
        //__syncthreads();
        result[idx] = ret;
    }
}

bool check(float *img, float *kernel, float *result, int n, int m, int kw, int kh, int out_n, int out_m, bool padding){
    for(int i = 0; i < out_n; i++){
        for(int j = 0; j < out_m; j++){
            float cur = 0.0;
            for(int p = 0; p < kw; p++){
                for(int q = 0; q < kh; q++){
                    //cur += img[(i + p) * m + (j + q)] * kernel[p * kh + q];
                    int cur_x = 0, cur_y = 0;
                    if(padding == true){
                        cur_x = i - kw /2 + p;
                        cur_y = j - kh /2 + q;
                    }
                    else{
                        cur_x = i + p;
                        cur_y = j + q;
                    }
                    if(cur_x >= 0 and cur_x < n and cur_y >= 0 and cur_y < m){
                        cur += img[cur_x * m + cur_y] * kernel[p * kh + q];
                    }
                }
            }
            //printf("%f %f\n", cur, result[i * out_m + j]);
            //printf("%f\n", cur);
            if(abs(cur - result[i * out_m + j]) > eps){
                cout<<cur<<' '<<result[i * out_m + j]<<endl;
                cout<<"Not Equal !!!"<<endl;
                exit(0);
            }
            //cout<<endl;
        }
    }
    cout<<"Nice !!! Equal!!"<<endl;
    return true;
}

int main(){
    bool padding = false; 
    int n = 512;
    int m = 512;
    int kh = 3;
    int kw = 3;
    int out_n = 0, out_m = 0;

    if(padding == false){
        out_n = (n - kw + 1);
        out_m = (m - kh + 1);
    }
    else{
        out_n = n;
        out_m = m;
    }
    
    size_t sizer = sizeof(float);
    float *kernel = NULL;
    kernel = (float*)malloc(kw * kh * sizer);

    for(int i = 0; i < kw; i++){
        for(int j = 0; j < kh; j++){
            kernel[i * kh + j] = 1;
        }
    }

    float *img = NULL;
    img = (float*)malloc(n * m * sizer);

    for(int i = 0; i < n; i++){
        for(int j = 0; j < m; j++){
            img[i * m + j] = (i + j) % 256;
            //cout<<img[i * m + j]<<' ';
        }
        //cout<<endl;
    }
    
    float *result = (float*)malloc(out_m * out_n * sizer);

    float *img_d = NULL;
    float *kernel_d = NULL;
    float *result_d = NULL;
    hipMalloc((void**)&kernel_d, kh * kw * sizer);
    hipMalloc((void**)&img_d, n * m * sizer);
    hipMalloc((void**)&result_d, out_m * out_n * sizer);

    hipMemcpy(img_d, img, n * m * sizer, hipMemcpyHostToDevice);
    hipMemcpy(kernel_d, kernel, kh * kw * sizer, hipMemcpyHostToDevice);
    
    dim3 threadPerBlock(2, 2);
    dim3 BlockPerGrid((out_n + threadPerBlock.x - 1) / threadPerBlock.x, (out_m + threadPerBlock.y - 1)/threadPerBlock.y);

    convolution2d<<<BlockPerGrid, threadPerBlock>>>(img_d, kernel_d, result_d, n, m,  kw, kh, out_n, out_m, padding);

    hipDeviceSynchronize();
    hipMemcpy(result, result_d, out_n * out_m * sizer, hipMemcpyDeviceToHost);
    
    // for(int i = 0; i < out_n; i++){
    //     for(int j = 0; j < out_m; j++){
    //         cout<<result[i * out_m + j]<<' ';
    //     }
    //     cout<<endl;
    // }

    check(img, kernel, result, n, m, kw, kh, out_m, out_n, padding);

    free(img);
    free(kernel);
    free(result);

    hipFree(img_d);
    hipFree(kernel_d);
    hipFree(result_d);

    return 0;

}